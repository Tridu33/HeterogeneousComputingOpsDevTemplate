#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
__global__ void kernel4array_mul_scalar (double *vec, double scalar, int num_elements) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elements) {
    vec[idx] = vec[idx] * scalar;
  }
}


void run_kernel4array_multiply_with_scalar(double *vec, double scalar, int num_elements) {
  dim3 dimBlock(256, 1, 1);
  dim3 dimGrid(ceil((double)num_elements / dimBlock.x));

  kernel4array_mul_scalar<<<dimGrid, dimBlock>>>(vec, scalar, num_elements); // 返回值保存在vec

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::stringstream strstr;
    strstr << "run_kernel4array_mul_scalar launch failed" << std::endl;
    strstr << "dimBlock: " << dimBlock.x << ", " << dimBlock.y << std::endl;
    strstr << "dimGrid: " << dimGrid.x << ", " << dimGrid.y << std::endl;
    strstr << hipGetErrorString(error);
    throw strstr.str();
  }
}
