#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CEIL(a, b) ((a + b - 1) / (b))
#define FLOAT4(a) *(float4 *)(&(a))

#define cudaCheck(err) _cudaCheck(err, __FILE__, __LINE__)
void _cudaCheck(hipError_t error, const char *file, int line) {
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s(line %d):\n%s\n", file, line, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    return;
};

// 核函数，纯纯放进去GPU执行的逻辑
__global__ void kernel4elementwise_add_float4(float* a, float* b, float* c, int N) {
    int idx = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
    if (idx >= N) return;
    
    float4 tmp_a = FLOAT4(a[idx]);
    float4 tmp_b = FLOAT4(b[idx]);
    float4 tmp_c;
    tmp_c.x = tmp_a.x + tmp_b.x;
    tmp_c.y = tmp_a.y + tmp_b.y;
    tmp_c.z = tmp_a.z + tmp_b.z;
    tmp_c.w = tmp_a.w + tmp_b.w;
    FLOAT4(c[idx]) = tmp_c;
}

// 1. host数据传进去核函数
// 2. <<<调用核函数计算>>>
// 3. GPU数据指针c_device，把计算结果返回host数据指针c_host
void run_kernel4elementwise_add_float4(float* a_h,
                                       float* b_h,
                                       float* c_h,
                                       int N) {
  float* a_d = nullptr;
  float* b_d = nullptr;
  float* c_d = nullptr;
  cudaCheck(hipMalloc((void**)&a_d, N * sizeof(float)));
  cudaCheck(hipMalloc((void**)&b_d, N * sizeof(float)));
  cudaCheck(hipMalloc((void**)&c_d, N * sizeof(float)));
  cudaCheck(hipMemcpy(a_d, a_h, N * sizeof(float), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(b_d, b_h, N * sizeof(float), hipMemcpyHostToDevice));
  int block_size = 1024;
  int grid_size = CEIL(CEIL(N, 4), 1024);
  kernel4elementwise_add_float4<<<grid_size, block_size>>>(a_d, b_d, c_d, N);
  // D2H 返回值保存在c_h
  cudaCheck(hipMemcpy(c_h, c_d, N * sizeof(float), hipMemcpyDeviceToHost));
}
